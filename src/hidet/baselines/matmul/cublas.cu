#include <cassert>
#include <iostream>
#include <hipblas.h>
#include <hidet/packedfunc.h>


#define CUBLAS_CALL(func) {                                     \
    hipblasStatus_t e = (func);                                  \
    if(e != HIPBLAS_STATUS_SUCCESS) {                                      \
        std::cerr << __FILE__ << ": " << __LINE__ << ":"        \
        << "CUBLAS: error code " << e << std::endl;             \
    }}                                                          \


// live with the program, leaving the destroying to the driver
static hipblasHandle_t cublas_handle = nullptr;


static hipError_t cublas_sgemm(int M, int N, int K, float const *A, float const *B, float *C) {
    if(cublas_handle == nullptr) {
        CUBLAS_CALL(hipblasCreate(&cublas_handle));
    }
    float alpha = 1.0;
    float beta = 0.0;
    CUBLAS_CALL(hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, A,
                         M, B, K, &beta, C, M));

    return hipSuccess;
}

/*
 * params: N, M, K, A, B, C
 */
DLL void MatmulCublas(int num_args, int *arg_types, void **args) {
    assert(num_args == 6);
    assert(arg_types[0] == INT32);
    int M = *static_cast<int *>(args[0]);
    assert(arg_types[1] == INT32);
    int N = *static_cast<int *>(args[1]);
    assert(arg_types[2] == INT32);
    int K = *static_cast<int *>(args[2]);
    assert(arg_types[3] == FLOAT32);
    auto *A = static_cast<float *>(args[3]);
    assert(arg_types[4] == FLOAT32);
    auto *B = static_cast<float *>(args[4]);
    assert(arg_types[5] == FLOAT32);
    auto *C = static_cast<float *>(args[5]);

    cublas_sgemm(M, N, K, A, B, C);
}

